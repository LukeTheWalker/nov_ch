
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <vector>
#include <utility>
#include <fstream>
#include <algorithm>

#define LOCAL_QUEUE_SIZE 128
#define PERSONAL_QUEUE_SIZE 64
#define LWS 512

using namespace std;
using Graph = vector<vector<int> >;

__host__ __device__ int round_div_up (int a, int b){
    return (a + b - 1)/b;
}

void cuda_err_check (hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        fprintf (stderr, "CUDA error: %s (%s:%d)\n", hipGetErrorString (err), file, line);
        exit (EXIT_FAILURE);
    }
}

__global__ void kernel (
    int numNodes, 
    int *d_nodePtrs, int *d_nodeNeighbors, 
    int *d_currLevelNodes, int *d_nodeVisited, const int numCurrLevelNodes,
    int4 *d_nextLevelNodes4, int *numNextLevelNodes
    ){
    extern __shared__ int4 lmem[];
    int * local_queue = ((int*)lmem) + 1;
    int * local_queue_size = ((int*)lmem);
    if (threadIdx.x == 0) *local_queue_size = 0;
    
    __syncthreads();

    int personal_queue_size = 0;
    int4 personal_queue4[PERSONAL_QUEUE_SIZE/4];

    int *personal_queue = (int*)personal_queue4;
    int *d_nextLevelNodes = (int*)d_nextLevelNodes4;


    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < numCurrLevelNodes){
        int node = d_currLevelNodes[tid];
        int start = d_nodePtrs[node];
        int end = d_nodePtrs[node + 1];
        for (int i = start; i < end; i++){
            int neighbor = d_nodeNeighbors[i];
            if (d_nodeVisited[neighbor] == 0 && (atomicCAS(&d_nodeVisited[neighbor], 0, 1)) == 0){
                int index = -1;
                if (personal_queue_size < PERSONAL_QUEUE_SIZE){
                    personal_queue[personal_queue_size++] = neighbor;
                    continue;
                }
                if (*local_queue_size < LOCAL_QUEUE_SIZE){
                    if ((index = atomicAdd_block(local_queue_size, 1)) < LOCAL_QUEUE_SIZE){
                        local_queue[index] = neighbor;
                        continue;
                    }
                }
                d_nextLevelNodes[atomicAdd(numNextLevelNodes, 1)] = neighbor;
            }
        }
    }
    
    // merge personal queue with local queue
    // int local_index = atomicAdd(local_queue_size, personal_queue_size);
    // for (int i = 0; i < personal_queue_size; i++)
    //     local_queue[local_index + i] = personal_queue[i];


    // merge personal queue with global queue
    // int local_index = atomicAdd(numNextLevelNodes, personal_queue_size);
    // for (int i = 0; i < personal_queue_size; i++)
    //     d_nextLevelNodes[local_index + i] = personal_queue[i];

    //merge personal queue with global queue vectorized
    int local_index = atomicAdd(numNextLevelNodes, personal_queue_size);
    if (personal_queue_size > 0){
        int new_personal_queue_size = personal_queue_size;
        int local_offset = local_index%4 == 0 ? 0 : 4-local_index%4;
        if (local_offset > personal_queue_size){
            for (int i = 0; i < personal_queue_size; i++)
                d_nextLevelNodes[local_index + i] = personal_queue[i];
        }
        else{
                // printf("Misaligned local_index: %d\n", local_index);
            for (int i = 0; i < local_offset; i++)
                d_nextLevelNodes[local_index + i] = personal_queue[personal_queue_size-1-i];
            local_index = local_index + local_offset;
            new_personal_queue_size = personal_queue_size - local_offset;
        
            int nquart_personal_queue_size = (new_personal_queue_size - new_personal_queue_size%4)/4;

            for (int i = 0; i < nquart_personal_queue_size; i++)
                d_nextLevelNodes4[local_index/4 + i] = personal_queue4[i];

            for (int i = nquart_personal_queue_size * 4; i < new_personal_queue_size; i++)
                d_nextLevelNodes[local_index + i] = personal_queue[i];
            }
        }

    __syncthreads();
    // merge local queue with global queue
    int number_of_nodes_to_merge_per_thread = *local_queue_size / blockDim.x;
    int start_index = threadIdx.x * number_of_nodes_to_merge_per_thread;
    int end_index = start_index + number_of_nodes_to_merge_per_thread;
    if (threadIdx.x == blockDim.x - 1) end_index = *local_queue_size;
    local_index = atomicAdd(numNextLevelNodes, end_index - start_index);
    for (int i = 0; i < end_index - start_index; i++)
        d_nextLevelNodes[local_index + i] = local_queue[start_index + i];
    
}

void kernel_launch (
    int numNodes, 
    int *d_nodePtrs, int *d_nodeNeighbors, 
    int *d_currLevelNodes, int * numCurrentLevelNodes, 
    int *d_nodeVisited, int lws = LWS
    ){

    int numBlocks;
    hipError_t err;

    int4 *d_nextLevelNodes;
    int *numNextLevelNodes;
    
    int * h_currentLevelNodes;
    err = hipHostMalloc((void**)&h_currentLevelNodes, numNodes * sizeof(int), hipHostMallocDefault); cuda_err_check(err, __FILE__, __LINE__);

    int * h_numCurrentLevelNodes = (int*)malloc(sizeof(int));
    err = hipMemcpy(h_numCurrentLevelNodes, numCurrentLevelNodes, sizeof(int), hipMemcpyDeviceToHost); cuda_err_check(err, __FILE__, __LINE__);

    int nquarts_nodes = round_div_up(numNodes, 4);
    err = hipMalloc((void**)&numNextLevelNodes, sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMalloc((void**)&d_nextLevelNodes, nquarts_nodes * sizeof(int4)); cuda_err_check(err, __FILE__, __LINE__);
    
    err = hipMemset(numNextLevelNodes, 0, sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);

    int level = 0;

    hipEvent_t start, stop;

    err = hipEventCreate(&start); cuda_err_check(err, __FILE__, __LINE__);
    err = hipEventCreate(&stop); cuda_err_check(err, __FILE__, __LINE__);

    err = hipEventRecord(start); cuda_err_check(err, __FILE__, __LINE__);
    err = hipEventRecord(stop); cuda_err_check(err, __FILE__, __LINE__);
    err = hipEventSynchronize(stop); cuda_err_check(err, __FILE__, __LINE__);

    float milliseconds = 0;
    err = hipEventElapsedTime(&milliseconds, start, stop); cuda_err_check(err, __FILE__, __LINE__);

    cout << "Time used before kernel launch: " << milliseconds << " ms" << endl;

    float total_time = 0;

    while (*h_numCurrentLevelNodes > 0){

        numBlocks = round_div_up(*h_numCurrentLevelNodes, lws);

        err = hipMemset(numNextLevelNodes, 0, sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMemset(d_nextLevelNodes, 0, nquarts_nodes * sizeof(int4)); cuda_err_check(err, __FILE__, __LINE__);


        // cout << "Launching kernel with " << numBlocks << " blocks and " << lws << " threads per block" << endl;
        err = hipEventRecord(start); cuda_err_check(err, __FILE__, __LINE__);
        kernel<<<numBlocks, lws, sizeof(int)*(LOCAL_QUEUE_SIZE+1)>>>(numNodes, d_nodePtrs, d_nodeNeighbors, d_currLevelNodes, d_nodeVisited, *h_numCurrentLevelNodes, d_nextLevelNodes, numNextLevelNodes);
        err = hipEventRecord(stop); cuda_err_check(err, __FILE__, __LINE__);
        err = hipEventSynchronize(stop); cuda_err_check(err, __FILE__, __LINE__);

        err = hipDeviceSynchronize(); cuda_err_check(err, __FILE__, __LINE__);

        err = hipMemcpy(numCurrentLevelNodes, numNextLevelNodes, sizeof(int), hipMemcpyDeviceToDevice); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMemcpy(d_currLevelNodes, d_nextLevelNodes, numNodes * sizeof(int), hipMemcpyDeviceToDevice); cuda_err_check(err, __FILE__, __LINE__);

        int old_numCurrentLevelNodes = *h_numCurrentLevelNodes;

        err = hipMemcpy(h_numCurrentLevelNodes, numCurrentLevelNodes, sizeof(int), hipMemcpyDeviceToHost); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMemcpy(h_currentLevelNodes, d_currLevelNodes, *h_numCurrentLevelNodes * sizeof(int), hipMemcpyDeviceToHost); cuda_err_check(err, __FILE__, __LINE__);

        sort(h_currentLevelNodes, h_currentLevelNodes + *h_numCurrentLevelNodes);

        float milliseconds = 0;
        err = hipEventElapsedTime(&milliseconds, start, stop); cuda_err_check(err, __FILE__, __LINE__);
        cout << "Time taken for kernel execution: " << milliseconds << " ms " << "with " << numBlocks << " blocks and " << lws << " threads per block" << " at level " << level << " with " << old_numCurrentLevelNodes << " nodes" << " generated " << *h_numCurrentLevelNodes << " nodes" << endl;
        level++;
        total_time += milliseconds;

        // cout << "numCurrentLevelNodes on CPU: " << *h_numCurrentLevelNodes << endl;
        // cout << "Level " << level << " on CPU: ";
        // for (int i = 0; i < *h_numCurrentLevelNodes; i++){
        //     cout << h_currentLevelNodes[i] << " ";
        // }
        // cout << endl;
    }

    cout << "Total time taken for kernel execution: " << total_time << " ms" << endl;

    err = hipFree(numNextLevelNodes); cuda_err_check(err, __FILE__, __LINE__);
    err = hipFree(d_nextLevelNodes); cuda_err_check(err, __FILE__, __LINE__);

    err = hipHostFree(h_currentLevelNodes); cuda_err_check(err, __FILE__, __LINE__);

    err = hipEventDestroy(start); cuda_err_check(err, __FILE__, __LINE__);
    err = hipEventDestroy(stop); cuda_err_check(err, __FILE__, __LINE__);
}

int main (int argc, char ** argv){
    if (argc != 2){
        cout << "Usage: " << argv[0] << " <graph_file>" << endl;
        exit(1);
    }

    int numNodes, numEdges;
    hipError_t err;

    int *d_nodePtrs;
    int *d_nodeNeighbors;

    {
        Graph g;
        ifstream infile;
        infile.open(argv[1]);
        infile >> numNodes >> numEdges;
        g.resize(numNodes);
        for (int i = 0; i < numEdges; i++){
            int src, dst;
            infile >> src >> dst;
            g[src].push_back(dst);
            g[dst].push_back(src);
        }
        infile.close();

        err = hipMalloc((void**)&d_nodePtrs, (numNodes+1) * sizeof(int*)); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMalloc((void**)&d_nodeNeighbors, numEdges * 2 * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);
     
        int * h_nodePtrs = (int*)malloc((numNodes+1) * sizeof(int));
        int * h_nodeNeighbors = (int*)malloc(numEdges * 2 * sizeof(int));

        int ptr = 0;
        for (int i = 0; i < numNodes; i++){
            h_nodePtrs[i] = ptr;
            for (int j = 0; j < g[i].size(); j++){
                h_nodeNeighbors[ptr] = g[i][j];
                ptr++;
            }
        }
        h_nodePtrs[numNodes] = ptr;

        err = hipMemcpy(d_nodePtrs, h_nodePtrs, (numNodes+1) * sizeof(int), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMemcpy(d_nodeNeighbors, h_nodeNeighbors, numEdges * 2 * sizeof(int), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);
    }

    int *d_currLevelNodes;
    int *numCurrLevelNodes;
    int *d_nodeVisited;

    err = hipMalloc((void**)&d_currLevelNodes, numNodes * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMalloc((void**)&d_nodeVisited, numNodes * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMalloc((void**)&numCurrLevelNodes, sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);
    
    err = hipMemset(d_currLevelNodes, 0, numNodes * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMemset(d_nodeVisited, 0, numNodes * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);

    int number_of_current_level_nodes = 1;
    int startNode = 0;
    char visited = 1;

    err = hipMemcpy(numCurrLevelNodes, &number_of_current_level_nodes, sizeof(int), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMemcpy(d_currLevelNodes, &startNode, sizeof(int), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMemcpy(&d_nodeVisited[startNode], &visited, sizeof(char), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);

    kernel_launch(numNodes, d_nodePtrs, d_nodeNeighbors, d_currLevelNodes, numCurrLevelNodes, d_nodeVisited);

    err = hipFree(d_nodePtrs); cuda_err_check(err, __FILE__, __LINE__);
    err = hipFree(d_nodeNeighbors); cuda_err_check(err, __FILE__, __LINE__);
    err = hipFree(d_currLevelNodes); cuda_err_check(err, __FILE__, __LINE__);
    err = hipFree(d_nodeVisited); cuda_err_check(err, __FILE__, __LINE__);
    err = hipFree(numCurrLevelNodes); cuda_err_check(err, __FILE__, __LINE__);

    return 0;
}