
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <vector>
#include <utility>
#include <fstream>
#include <algorithm>

using namespace std;
using Graph = vector<vector<int> >;

__host__ __device__ int round_div_up (int a, int b){
    return (a + b - 1)/b;
}

void cuda_err_check (hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        fprintf (stderr, "CUDA error: %s (%s:%d)\n", hipGetErrorString (err), file, line);
        exit (EXIT_FAILURE);
    }
}

__global__ void kernel (
    int numNodes, 
    int *d_nodePtrs, int *d_nodeNeighbors, 
    int *d_currLevelNodes, int *d_nodeVisited, int * numCurrLevelNodes,
    int *d_nextLevelNodes, int *numNextLevelNodes
    ){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < *numCurrLevelNodes){
        int node = d_currLevelNodes[tid];
        int start = d_nodePtrs[node];
        int end = d_nodePtrs[node + 1];
        for (int i = start; i < end; i++){
            int neighbor = d_nodeNeighbors[i];
            if (d_nodeVisited[neighbor] == 0 && (atomicCAS(&d_nodeVisited[neighbor], 0, 1)) == 0){
                int index = atomicAdd(numNextLevelNodes, 1);
                d_nextLevelNodes[index] = neighbor;
            }
        }
    }
}

void kernel_launch (
    int numNodes, 
    int *d_nodePtrs, int *d_nodeNeighbors, 
    int *d_currLevelNodes, int * numCurrentLevelNodes, 
    int *d_nodeVisited, int lws = 256
    ){

    int numBlocks;
    hipError_t err;

    int *d_nextLevelNodes;
    int *numNextLevelNodes;
    char *something_changed;
    
    int * h_currentLevelNodes;
    err = hipHostMalloc((void**)&h_currentLevelNodes, numNodes * sizeof(int), hipHostMallocDefault); cuda_err_check(err, __FILE__, __LINE__);

    int * h_numCurrentLevelNodes = (int*)malloc(sizeof(int));
    err = hipMemcpy(h_numCurrentLevelNodes, numCurrentLevelNodes, sizeof(int), hipMemcpyDeviceToHost); cuda_err_check(err, __FILE__, __LINE__);

    err = hipMalloc((void**)&numNextLevelNodes, sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMalloc((void**)&d_nextLevelNodes, numNodes * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMalloc((void**)&something_changed, sizeof(char)); cuda_err_check(err, __FILE__, __LINE__);
    
    err = hipMemset(numNextLevelNodes, 0, sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);

    int level = 0;
    cout << "Level " << level << ": 0" << endl;
    level++;

    hipEvent_t start, stop;

    err = hipEventCreate(&start); cuda_err_check(err, __FILE__, __LINE__);
    err = hipEventCreate(&stop); cuda_err_check(err, __FILE__, __LINE__);

    float total_time = 0;

    while (*h_numCurrentLevelNodes > 0){

        numBlocks = round_div_up(*h_numCurrentLevelNodes, lws);

        err = hipMemset(numNextLevelNodes, 0, sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMemset(d_nextLevelNodes, 0, numNodes * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMemset(something_changed, 0, sizeof(char)); cuda_err_check(err, __FILE__, __LINE__);

        err = hipEventRecord(start); cuda_err_check(err, __FILE__, __LINE__);

        // cout << "Launching kernel with " << numBlocks << " blocks and " << lws << " threads per block" << endl;
        kernel<<<numBlocks, lws>>>(numNodes, d_nodePtrs, d_nodeNeighbors, d_currLevelNodes, d_nodeVisited, numCurrentLevelNodes, d_nextLevelNodes, numNextLevelNodes);

        err = hipEventRecord(stop); cuda_err_check(err, __FILE__, __LINE__);
        err = hipEventSynchronize(stop); cuda_err_check(err, __FILE__, __LINE__);

        err = hipDeviceSynchronize(); cuda_err_check(err, __FILE__, __LINE__);

        err = hipMemcpy(numCurrentLevelNodes, numNextLevelNodes, sizeof(int), hipMemcpyDeviceToDevice); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMemcpy(d_currLevelNodes, d_nextLevelNodes, numNodes * sizeof(int), hipMemcpyDeviceToDevice); cuda_err_check(err, __FILE__, __LINE__);

        err = hipMemcpy(h_numCurrentLevelNodes, numCurrentLevelNodes, sizeof(int), hipMemcpyDeviceToHost); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMemcpy(h_currentLevelNodes, d_currLevelNodes, *h_numCurrentLevelNodes * sizeof(int), hipMemcpyDeviceToHost); cuda_err_check(err, __FILE__, __LINE__);

        sort(h_currentLevelNodes, h_currentLevelNodes + *h_numCurrentLevelNodes);

        // cout << "numCurrentLevelNodes: " << *h_numCurrentLevelNodes << endl;
        // cout << "Level " << level << ": ";
        // for (int i = 0; i < *h_numCurrentLevelNodes; i++){
        //     cout << h_currentLevelNodes[i] << " ";
        // }
        // cout << endl;
        float milliseconds = 0;
        err = hipEventElapsedTime(&milliseconds, start, stop); cuda_err_check(err, __FILE__, __LINE__);
        cout << "Time taken for kernel execution: " << milliseconds << " ms " << "with " << numBlocks << " blocks and " << lws << " threads per block" << " at level " << level << " with " << *h_numCurrentLevelNodes << " nodes" << endl;
        level++;
        total_time += milliseconds;
    }

    cout << "Total time taken for kernel execution: " << total_time << " ms" << endl;

    err = hipFree(numNextLevelNodes); cuda_err_check(err, __FILE__, __LINE__);
    err = hipFree(d_nextLevelNodes); cuda_err_check(err, __FILE__, __LINE__);
    err = hipFree(something_changed); cuda_err_check(err, __FILE__, __LINE__);

    err = hipHostFree(h_currentLevelNodes); cuda_err_check(err, __FILE__, __LINE__);
}

int main (int argc, char ** argv){
    if (argc != 2){
        cout << "Usage: " << argv[0] << " <graph_file>" << endl;
        exit(1);
    }

    int numNodes, numEdges;
    hipError_t err;

    int *d_nodePtrs;
    int *d_nodeNeighbors;

    {
        Graph g;
        ifstream infile;
        infile.open(argv[1]);
        infile >> numNodes >> numEdges;
        g.resize(numNodes);
        for (int i = 0; i < numEdges; i++){
            int src, dst;
            infile >> src >> dst;
            g[src].push_back(dst);
            g[dst].push_back(src);
        }
        infile.close();

        err = hipMalloc((void**)&d_nodePtrs, (numNodes+1) * sizeof(int*)); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMalloc((void**)&d_nodeNeighbors, numEdges * 2 * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);
     
        int * h_nodePtrs = (int*)malloc((numNodes+1) * sizeof(int));
        int * h_nodeNeighbors = (int*)malloc(numEdges * 2 * sizeof(int));

        int ptr = 0;
        for (int i = 0; i < numNodes; i++){
            h_nodePtrs[i] = ptr;
            for (int j = 0; j < g[i].size(); j++){
                h_nodeNeighbors[ptr] = g[i][j];
                ptr++;
            }
        }
        h_nodePtrs[numNodes] = ptr;

        err = hipMemcpy(d_nodePtrs, h_nodePtrs, (numNodes+1) * sizeof(int), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);
        err = hipMemcpy(d_nodeNeighbors, h_nodeNeighbors, numEdges * 2 * sizeof(int), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);err = hipMemcpy(&d_nodePtrs[numNodes], &ptr, sizeof(int), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);
    }

    int *d_currLevelNodes;
    int *numCurrLevelNodes;
    int *d_nodeVisited;

    err = hipMalloc((void**)&d_currLevelNodes, numNodes * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMalloc((void**)&d_nodeVisited, numNodes * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMalloc((void**)&numCurrLevelNodes, sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);
    
    err = hipMemset(d_currLevelNodes, 0, numNodes * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMemset(d_nodeVisited, 0, numNodes * sizeof(int)); cuda_err_check(err, __FILE__, __LINE__);

    int number_of_current_level_nodes = 1;
    int startNode = 0;
    char visited = 1;

    err = hipMemcpy(numCurrLevelNodes, &number_of_current_level_nodes, sizeof(int), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMemcpy(d_currLevelNodes, &startNode, sizeof(int), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMemcpy(&d_nodeVisited[startNode], &visited, sizeof(char), hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);

    kernel_launch(numNodes, d_nodePtrs, d_nodeNeighbors, d_currLevelNodes, numCurrLevelNodes, d_nodeVisited);

    err = hipFree(d_nodePtrs); cuda_err_check(err, __FILE__, __LINE__);
    err = hipFree(d_nodeNeighbors); cuda_err_check(err, __FILE__, __LINE__);
    err = hipFree(d_currLevelNodes); cuda_err_check(err, __FILE__, __LINE__);
    err = hipFree(d_nodeVisited); cuda_err_check(err, __FILE__, __LINE__);
    err = hipFree(numCurrLevelNodes); cuda_err_check(err, __FILE__, __LINE__);

    return 0;
}